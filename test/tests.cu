#include <tuple>

#define CATCH_CONFIG_MAIN
#include "catch.hpp"
#include <stdio.h>
#include "../src/lib.h"

#define HANDLE_ERROR(status) { cudaEvalStatus((status),__FILE__,__LINE__); }

static inline void cudaEvalStatus(hipError_t status, const char * filename, unsigned int line){
    if (status != hipSuccess){
        printf("CUDA error in %s (line %i): %s\n",filename,line,hipGetErrorString(status));
        hipDeviceReset();
        exit(0);
    }
}

TEST_CASE("Ex1", "[example]")
{
    int *d_v1, *d_v2;    
    int *v1, *v2, *v3;
    v1 = (int*)malloc(sizeof(int)*2);
    v2 = (int*)malloc(sizeof(int)*2);
    v3 = (int*)malloc(sizeof(int)*2);
    v1[0]=v1[1]=1;
    v2[0]=v2[1]=2;
    hipMalloc(&d_v1, 2*sizeof(int)); 
    hipMalloc(&d_v2, 2*sizeof(int));
    hipMemcpy(d_v1, v1, 2*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_v2, v2, 2*sizeof(int), hipMemcpyHostToDevice);
    HANDLE_ERROR(hipPeekAtLastError());
    soma<<<2,1>>>(2,d_v1,d_v2);
    hipMemcpy(v3, d_v2, 2*sizeof(int), hipMemcpyDeviceToHost);
    for(int i=0;i<2;i++){
        printf("%d: %d==%d+%d\n", i, v3[i], v1[i], v2[i]);
        REQUIRE(v3[i]==v1[i]+v2[i]);
    }
    hipFree(d_v1);
    hipFree(d_v2);
    free(v1);
    free(v2);
    free(v3);
}
