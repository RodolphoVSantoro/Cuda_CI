#include "hip/hip_runtime.h"
#include "lib.h"

__global__
void soma(int N, int *v1, int *v2){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N)
    v2[i]+=v1[i];
}
