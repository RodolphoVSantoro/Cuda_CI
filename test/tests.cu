#include <tuple>

#define CATCH_CONFIG_MAIN
#include "catch.hpp"

#include "../src/lib.h"

TEST_CASE("Ex1", "[example]")
{
    int *d_v1, *d_v2;
    int v1[2] = {1,1},
    v2[2] = {2,2}, v3[2];
    hipMalloc(&d_v1, 2*sizeof(int)); 
    hipMalloc(&d_v2, 2*sizeof(int));
    hipMemcpy(d_v1, v1, 2*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_v2, v2, 2*sizeof(int), hipMemcpyHostToDevice);
    soma<<<2,1>>>(2,d_v1,d_v2);
    hipMemcpy(v3, d_v2, 2*sizeof(int), hipMemcpyDeviceToHost);
    for(int i=0;i<2;i++){
        REQUIRE(v3[i]==v1[i]+v2[i]);
    }
    hipFree(d_v1);
    hipFree(d_v2);
}
