#include "hip/hip_runtime.h"
#include "lib.h"
#include <stdio.h>

__global__ void soma(int N, int *v1, int *v2){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  printf("blockid=%d, blockdim=%d, threadid=%d\n", blockIdx.x, blockDim.x, threadIdx.x);
  printf("i=%d, v1=%d, v2=%d\n", i, v1[i], v2[i]);
  if(i<N)
    v2[i]+=v1[i];
}
